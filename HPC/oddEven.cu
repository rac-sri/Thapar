#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 20
#include <time.h>
#include <stdlib.h>
#include<iostream>
#define intswap(A,B) {int temp=A;A=B;B=temp;}
using namespace std;
__global__ void sort(int *c,int *count)
{
    int l;
    if(*count%2==0)
          l=*count/2;
    else
         l=(*count/2)+1;
    for(int i=0;i<l;i++)
    {
            if((!(threadIdx.x&1)) && (threadIdx.x<(*count-1)))  //even phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }
            __syncthreads();
            if((threadIdx.x&1) && (threadIdx.x<(*count-1)))     //odd phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }
            __syncthreads();
    }
}
int main()
{
 clock_t tStart = clock();
 int a[N],b[N],n;
 n=10;
 srand(time(NULL)); 
 for(int i=0;i<n;i++) {
     int r= rand()%10;
     a[i]=r;
 }
  printf("Array Before Sorting: \n");
  for(int i=0;i<n;i++)
          {
          printf("%d ",a[i]);
          }
  int *c,*count;
  hipMalloc((void**)&c,sizeof(int)*N);
  hipMalloc((void**)&count,sizeof(int));
  hipMemcpy(c,&a,sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(count,&n,sizeof(int),hipMemcpyHostToDevice);
  sort<<< 1,n >>>(c,count);
  hipMemcpy(&b,c,sizeof(int)*N,hipMemcpyDeviceToHost);
  printf("\nSorted Array using Cuda C : \n");
  for(int i=0;i<n;i++)
      {
         printf("%d ",b[i]);
      }
  printf("\n");
 cout<<"Execution Time:"<<(double)(clock() - tStart)/CLOCKS_PER_SEC;
}

