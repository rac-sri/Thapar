#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 20
#include <time.h>
#include <stdlib.h>
#include <fstream>
#include <string.h>
#include<iostream>
#define intswap(A,B) {int temp=A;A=B;B=temp;}
using namespace std;
__global__ void sort(int *c,int *count)
{
    int l;
    if(*count%2==0)
          l=*count/2;
    else
         l=(*count/2)+1;
    for(int i=0;i<l;i++)
    {
            if((!(threadIdx.x&1)) && (threadIdx.x<(*count-1)))  //even 
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }
            __syncthreads();
            if((threadIdx.x&1) && (threadIdx.x<(*count-1)))     //odd 
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                  intswap(c[threadIdx.x], c[threadIdx.x+1]);
            }
            __syncthreads();
    }
}

void getData(char buff[],int *a,int &i) 
{ 
		
   char *token = strtok(buff,","); 
	//cout<<token<<endl;
   int counter=0; 
		
	
		  
   while( token != NULL&&i<N )  
   { 
 counter++; 
 //cout<<token<<" ";
			a[i]=atoi(token);
			 i++;
// fgets(buff, N, (FILE*)fp); 
	 count++; 
…printf( " %s\n",token); 
      token = strtok(NULL,","); 
			 

   }	 
	//cout<<endl;  
} 


void fill(int *a){
		
  FILE *fp = fopen("data2.csv", "r");
      int count=0; 
			int i=0;
	do 
	{ 
	 char buff[1000000]; 
	 fgets(buff, N, (FILE*)fp); 
	 count++; 
	 if(count != 1) 
	 { 
	  //printf(buff);
		 //cout<<endl; 
		 if(i<N){
	 
	  getData(buff,a,i);
		} 
	 } 
	}while((getc(fp))!=EOF&&i<N);
}



int main()
{
    clock_t tStart = clock();
    int b[N],n;
   n=10;
     int *a;
 a=new int[N];
 fill(a);

 srand(time(NULL)); 

for(int x=0;x<n;x++){
    
}
  printf("Array Before Sorting: \n");
  for(int i=0;i<n;i++)
          {
          printf("%d ",a[i]);
          }
 
  int *c,*count;
  hipMalloc((void**)&c,sizeof(int)*N);
  hipMalloc((void**)&count,sizeof(int));
  hipMemcpy(c,&a,sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(count,&n,sizeof(int),hipMemcpyHostToDevice);
  sort<<< 1,n >>>(c,count);
  hipMemcpy(&b,c,sizeof(int)*N,hipMemcpyDeviceToHost);
  printf("\nSorted Array using Cuda C : \n");
  for(int i=0;i<n;i++)
      {
         printf("%d ",b[i]);
      }
  printf("\n");
 cout<<"Execution Time:"<<(double)(clock() - tStart)/CLOCKS_PER_SEC;
}

