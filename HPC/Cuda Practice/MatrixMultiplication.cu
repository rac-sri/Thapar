#include<stdio.h>
#include<hip/hip_runtime.h>
_global_ void c_hello(int* a1_d,int* a2_d,int* a3_d)
{
	a3_d[blockIdx.x]=0;
	//printf("\n%d y",*(a1_d));
	for(int i=0;i<3;i++)
	{
		a3_d[blockIdx.x]+=int((a1_d+3(blockIdx.x)+i))int((a2_d +3*(blockIdx.x) +i));
	}
	printf("%d \n",a3_d[blockIdx.x]);
	return;
}

int main()
{
	unsigned int n;
	scanf("%d",&n);
	int *arr1,*arr2,*a1_d,*a2_d,*a3_d;
	arr1=(int*)malloc(sizeof(int)*n*3);
	arr2=(int*)malloc(sizeof(int)*n*3);
	for(int i=0;i<n;i++)
	{
		int x,y,z;
		scanf("%d%d%d",&x,&y,&z);
		*(arr1+3*i)=x;
		*(arr1+i*3+1)=y;
		*(arr1+i*3+2)=z;
		scanf("%d%d%d",&x,&y,&z);
		*(arr2+i*3)=x;
		*(arr2+i*3+1)=y;
		*(arr2+i*3+2)=z;
	}
	hipMalloc((void**) &a1_d,sizeof(int)*n*3);
	hipMalloc((void**) &a2_d,sizeof(int)*n*3);
	hipMalloc((void**) &a3_d,sizeof(int)*n);
	hipMemcpy(a1_d,arr1,sizeof(int)*n*3,hipMemcpyHostToDevice);
	hipMemcpy(a2_d,arr2,sizeof(int)*n*3,hipMemcpyHostToDevice);
	//printf("cpu %d\n",*arr1);
	dim3 dd;
	dd={n};
	//hipMalloc((void**)&d_a, sizeof(int) * 1024);
	c_hello<<<dd,1>>>(a1_d,a2_d,a3_d);
	hipDeviceSynchronize();
	return 0;
}