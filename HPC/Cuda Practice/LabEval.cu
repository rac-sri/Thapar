
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
fstream fin;

__global__ void matMul(float *A,float *B,float *C,int n) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int idx=ROW*n+COL;

    if (ROW < n && COL < n) {
        C[idx]= A[idx ] * B[idx ];
    }
}

void matrixMultiplication(float *A, float *B, float *C, int N){
    
     dim3 threadsPerBlock(N, N);
     dim3 blocksPerGrid(1, 1);
      if (N*N > 512){
          threadsPerBlock.x = 512;
          threadsPerBlock.y = 512;
          blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
          blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
      }
      matMul<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}
int main(){
     fin.open("./dataFile.csv", ios::in);
     clock_t st1,e1,st,end;
     int N=8;
     cout<<N<<endl;
     int size= N*N;
     float *h_A,*h_B,*h_C; //host
     float *d_A,*d_B,*d_C; //device copy GPU

     h_A=(float *)malloc(size);
     h_B=(float *)malloc(size);
     h_C=(float *)malloc(size);
     hipMalloc((void **)&d_A, size);
     hipMalloc((void **)&d_B, size);
     hipMalloc((void **)&d_C, size);
     string s;
     fin>>s;
     stringstream str(s);
     for(int i=0; i<size; i++){
        getline(str,s,',');
        h_A[i] = stoi(s);
        cout<<h_A[i]<<" ";
     }
     cout<<endl;
     for(int i=0; i<size; i++){
        getline(str,s,',');
        h_B[i] = stoi(s);
        cout<<h_B[i]<<" ";
     }
    cout<<endl;
    st1=clock();
    for(int i=0; i<N; i++){
            for(int j=0; j<N; j++){
                 cout<< h_A[i*N+j]*h_B[i*N+j]<<" ";
            }
            cout<<endl;
        }
        
     e1=clock();
    
    
     double time_taken =((double)(e1-st1))/CLOCKS_PER_SEC;
     cout<<"\ncomputational time using sequential is "<<time_taken<<" secs\n";
     hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
     hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
    
     st=clock();
     matrixMultiplication(d_A, d_B, d_C, N);

     hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
     end=clock();
     cout<<endl;
     for(int i=0; i<N; i++){
         for(int j=0;j<N;j++)
         cout<<h_C[i*N+j]<<" ";
         cout<<endl;
     }
     
     double time_taken1 =((double)(end-st))/CLOCKS_PER_SEC;
     cout<<"\ncomputational time using cuda is "<<time_taken1<<" secs\n";
     fin.close();
     return 0; 
}